#include "CUDA_MC_Solver.hpp"
#include "CUDA_MC_LookBack_Solver.cuh"

CUDA_MC_Solver<LookBackOption>::CUDA_MC_Solver(int _Nseries, int _Nsteps)
{
    Nseries = _Nseries;
    Nsteps = _Nsteps; 

    returns = NULL;
    assets = NULL;
    payoffs = NULL;
}

CUDA_MC_Solver<LookBackOption>::~CUDA_MC_Solver()
{
    hiprandDestroyGenerator( gen );

    if( returns )
	hipFree( returns );
    if( assets )
	hipFree( assets );
    if( payoffs )
	hipFree( payoffs );
};

void CUDA_MC_Solver<LookBackOption>::operator()(LookBackOption *option)
{
    assert( returns );
    assert( assets );
    assert( payoffs );
    float *localPayoffs = new float[Nseries];
    assert( localPayoffs );
    float scale = (float)option->T/(float)Nsteps;

    float call = 1.0;
    if(option->put)
	call = -1.0;

    int threadsPerBlock = 256;
    int nBlocks = Nseries/threadsPerBlock;
    LookBackKernel<option->type><<<nBlocks, threadsPerBlock>>>(returns, 
					    assets, 
					    payoffs, 
					    (float)option->r, 
					    (float)option->S0, 
					    (float)option->K, 
					    (float) option->sigma*scale, 
					    (float) option->T,
					    call,
					    Nseries, 
					    Nsteps);
    
    hipDeviceSynchronize();
    hipMemcpy( (void *) localPayoffs, payoffs, sizeof(float)*Nseries, hipMemcpyDeviceToHost );

    // This ends all the calls to cuda, now just averaging over the payoffs
    double avg;
    for(int i = 0; i < Nseries; i++)
    {
	avg += (double) localPayoffs[i];
    }
    avg /= (double) Nseries;
    option->price = avg;
    delete[] localPayoffs;
};

void CUDA_MC_Solver<LookBackOption>::init()
{
    assert( hipMalloc( (void **) &returns, Nseries*Nsteps*sizeof(float) ) == hipSuccess);
    assert( hipMalloc( (void **) &assets, Nseries*sizeof(float) ) == hipSuccess);
    assert( hipMalloc( (void **) &payoffs, Nseries*sizeof(float) ) == hipSuccess);

    assert( hiprandCreateGenerator( &gen, HIPRAND_RNG_PSEUDO_MTGP32 ) == HIPRAND_STATUS_SUCCESS);
    assert( hiprandSetPseudoRandomGeneratorSeed( gen, 1234ULL ) == HIPRAND_STATUS_SUCCESS);
    assert( hiprandGenerateNormal( gen, returns, Nseries*Nsteps, 0.0, 1.0 ) == HIPRAND_STATUS_SUCCESS);
};

void CUDA_MC_Solver<LookBackOption>::free()
{
    hiprandDestroyGenerator( gen );

    if( returns )
	hipFree( returns );
	returns = NULL;
    if( assets )
	hipFree( assets );
	assets = NULL;
    if( payoffs )
	hipFree( payoffs );
	payoffs = NULL;
};
