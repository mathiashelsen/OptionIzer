#include "hip/hip_runtime.h"
#include "CUDA_MC_Euro_Solver.hpp"

__global__ void EuroKernel( float *_x, float *_assets, float *_payoffs,
    float r,
    float S0,
    float K,
    float sigma,
    float T,
    float call,
    int Nseries,
    int Nsteps )
{
    int pathIndex = threadIdx.x + blockIdx.x*blockDim.x;
    if( pathIndex >= Nseries )
	return;
    
    _assets[pathIndex] = S0;

    for(int i = 0; i < Nsteps; i++)
    {
	_assets[pathIndex] *= (1.0 + r + sigma*_x[pathIndex*Nsteps + i]);
    }
    float tmp = call*(_assets[pathIndex] - K)*exp(-r*T);
    _payoffs[pathIndex] = (tmp > 0.0) ? tmp : 0.0;

    __syncthreads();
}

CUDA_MC_Euro_Solver::CUDA_MC_Euro_Solver(int _Nseries, int _Nsteps)
{
    Nseries = _Nseries;
    Nsteps = _Nsteps; 

}

CUDA_MC_Euro_Solver::~CUDA_MC_Euro_Solver()
{
    };

void CUDA_MC_Euro_Solver::operator()(VanillaOption *option)
{
    assert( hipMalloc( (void **) &returns, Nseries*Nsteps*sizeof(float) ) == hipSuccess);
    assert( hipMalloc( (void **) &assets, Nseries*sizeof(float) ) == hipSuccess);
    assert( hipMalloc( (void **) &payoffs, Nseries*sizeof(float) ) == hipSuccess);

    assert( hiprandCreateGenerator( &gen, HIPRAND_RNG_PSEUDO_MTGP32 ) == HIPRAND_STATUS_SUCCESS);
    assert( hiprandSetPseudoRandomGeneratorSeed( gen, 1234ULL ) == HIPRAND_STATUS_SUCCESS);
    assert( hiprandGenerateNormal( gen, returns, Nseries*Nsteps, 0.0, 1.0 ) == HIPRAND_STATUS_SUCCESS);
    float *localPayoffs = new float[Nseries];
    assert( localPayoffs );
    float scale = (float)option->T/(float)Nsteps;

    float call = 1.0;
    if(option->put)
	call = -1.0;

    int threadsPerBlock = 256;
    int nBlocks = Nseries/threadsPerBlock;
    EuroKernel<<<nBlocks, threadsPerBlock>>>(returns, 
					    assets, 
					    payoffs, 
					    (float)option->r, 
					    (float)option->S0, 
					    (float)option->K, 
					    (float) option->sigma*scale, 
					    (float) option->T,
					    call,
					    Nseries, 
					    Nsteps);
    
    hipDeviceSynchronize();
    hipMemcpy( (void *) localPayoffs, payoffs, sizeof(float)*Nseries, hipMemcpyDeviceToHost );

    // This ends all the calls to cuda, now just averaging over the payoffs
    double avg;
    for(int i = 0; i < Nseries; i++)
    {
	avg += (double) localPayoffs[i];
    }
    avg /= (double) Nseries;
    option->price = avg;
    delete[] localPayoffs;
    hiprandDestroyGenerator( gen );
    hipFree( returns );
    hipFree( assets );

};
