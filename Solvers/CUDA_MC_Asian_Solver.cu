#include "hip/hip_runtime.h"
#include "CUDA_MC_Solver.hpp"

__global__ void AsianKernel(float *_x, float *_assets, float *_payoffs,
    float r,
    float S0,
    float K,
    float sigma,
    float T,
    float call,
    int Nseries,
    int Nsteps )
{
    int pathIndex = threadIdx.x + blockIdx.x*blockDim.x;
    if( pathIndex >= Nseries )
	return;
    
    _assets[pathIndex] = S0;
    float avg = 0.0;
    for(int i = 0; i < Nsteps; i++)
    {
	_assets[pathIndex] *= (1.0 + r + sigma*_x[pathIndex*Nsteps + i]);
	avg += _assets[pathIndex];
    }
    avg /= (float) Nsteps;

    float tmp = call*(avg - K)*exp(-r*T);
    _payoffs[pathIndex] = (tmp > 0.0f) ? tmp : 0.0f;

    __syncthreads();

}


CUDA_MC_Solver<AsianOption>::CUDA_MC_Solver(int _Nseries, int _Nsteps)
{
    Nseries = _Nseries;
    Nsteps = _Nsteps; 

    returns = NULL;
    assets = NULL;
    payoffs = NULL;
}

CUDA_MC_Solver<AsianOption>::~CUDA_MC_Solver()
{
    hiprandDestroyGenerator( gen );

    if( returns )
	hipFree( returns );
    if( assets )
	hipFree( assets );
    if( payoffs )
	hipFree( payoffs );
};

void CUDA_MC_Solver<AsianOption>::operator()(AsianOption *option)
{
    assert( returns );
    assert( assets );
    assert( payoffs );
    float *localPayoffs = new float[Nseries];
    assert( localPayoffs );
    float scale = (float)option->T/(float)Nsteps;

    float call = 1.0;
    if(option->put)
	call = -1.0;

    int threadsPerBlock = 256;
    int nBlocks = Nseries/threadsPerBlock;
    AsianKernel<<<nBlocks, threadsPerBlock>>>(returns, 
					    assets, 
					    payoffs, 
					    (float)option->r, 
					    (float)option->S0, 
					    (float)option->K, 
					    (float) option->sigma*scale, 
					    (float) option->T,
					    call,
					    Nseries, 
					    Nsteps);
    
    hipDeviceSynchronize();
    hipMemcpy( (void *) localPayoffs, payoffs, sizeof(float)*Nseries, hipMemcpyDeviceToHost );

    // This ends all the calls to cuda, now just averaging over the payoffs
    double avg;
    for(int i = 0; i < Nseries; i++)
    {
	avg += (double) localPayoffs[i];
    }
    avg /= (double) Nseries;
    option->price = avg;
    delete[] localPayoffs;
};

void CUDA_MC_Solver<AsianOption>::init()
{
    assert( hipMalloc( (void **) &returns, Nseries*Nsteps*sizeof(float) ) == hipSuccess);
    assert( hipMalloc( (void **) &assets, Nseries*sizeof(float) ) == hipSuccess);
    assert( hipMalloc( (void **) &payoffs, Nseries*sizeof(float) ) == hipSuccess);

    assert( hiprandCreateGenerator( &gen, HIPRAND_RNG_PSEUDO_MTGP32 ) == HIPRAND_STATUS_SUCCESS);
    assert( hiprandSetPseudoRandomGeneratorSeed( gen, 1234ULL ) == HIPRAND_STATUS_SUCCESS);
    assert( hiprandGenerateNormal( gen, returns, Nseries*Nsteps, 0.0, 1.0 ) == HIPRAND_STATUS_SUCCESS);
};

